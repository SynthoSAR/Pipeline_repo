#include "imageLoad.cuh"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

void loadImageToGPU(const cv::Mat& img, ImageData& imgData) {
    imgData.width = img.cols;
    imgData.height = img.rows;
    imgData.channels = img.channels();

    size_t imageSize = img.total() * img.elemSize(); 
    CHECK_CUDA_ERROR(hipMalloc(&imgData.d_image, imageSize));
    CHECK_CUDA_ERROR(hipMemcpy(imgData.d_image, img.data, imageSize, hipMemcpyHostToDevice));
}
